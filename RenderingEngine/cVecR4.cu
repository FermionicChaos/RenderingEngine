#include "cVecR4.h"

#include <hip/hip_runtime.h>
#include "hip/hip_runtime.h"
#include ""

cVecR4::cVecR4() {
	hipMalloc(&d_ptr, 4*sizeof(double));
	h_ptr = new double[4];
}

cVecR4::~cVecR4() {
	hipFree(d_ptr);
	delete[] h_ptr;
}

cVecR4::cVecR4(cVecR4 & inp) {
	hipMalloc(&d_ptr, 4*sizeof(double));
	h_ptr = new double[4];
	hipMemcpy(d_ptr, inp.d_ptr, 4*sizeof(double), hipMemcpyDeviceToDevice);
	hipMemcpy(h_ptr, d_ptr, 4 * sizeof(double), hipMemcpyDeviceToHost);
	metric = inp.metric;
	vec = inp.vec;
}

cVecR4 & cVecR4::operator=(cVecR4 & rhs) {
	hipMalloc(&(this->d_ptr), 4 * sizeof(double));
	this->h_ptr = new double[4];
	hipMemcpy(this->d_ptr, rhs.d_ptr, 4 * sizeof(double), hipMemcpyDeviceToDevice);
	hipMemcpy(this->h_ptr, this->d_ptr, 4 * sizeof(double), hipMemcpyDeviceToHost);
	this->metric = rhs.metric;
	this->vec = rhs.vec;
	return *this;
}

cVecR4::cVecR4(cVecR4 && inp) {
	hipMalloc(&d_ptr, 4 * sizeof(double));
	h_ptr = new double[4];
	hipMemcpy(d_ptr, inp.d_ptr, 4 * sizeof(double), hipMemcpyDeviceToDevice);
	hipMemcpy(h_ptr, d_ptr, 4 * sizeof(double), hipMemcpyDeviceToHost);
	metric = inp.metric;
	vec = inp.vec;

	hipFree(inp.d_ptr);
	delete[] h_ptr;
}

cVecR4 & cVecR4::operator=(cVecR4 && rhs) {
	hipMalloc(&(this->d_ptr), 4 * sizeof(double));
	this->h_ptr = new double[4];
	hipMemcpy(this->d_ptr, rhs.d_ptr, 4 * sizeof(double), hipMemcpyDeviceToDevice);
	hipMemcpy(this->h_ptr, rhs.d_ptr, 4 * sizeof(double), hipMemcpyDeviceToHost);
	this->metric = rhs.metric;
	this->vec = rhs.vec;

	hipFree(rhs.d_ptr);
	delete[] h_ptr;
	return *this;
}

cVecR4::cVecR4(vec_t v_type, metric_t m_type, double z0, double z1, double z2, double z3) {
	vec = v_type;
	metric = m_type;

	hipMalloc(&d_ptr, 4*sizeof(double));
	h_ptr = new double[4];
	h_ptr[0] = z0;
	h_ptr[1] = z1;
	h_ptr[2] = z2;
	h_ptr[3] = z3;

	hipMemcpy(d_ptr, h_ptr, 4*sizeof(double), hipMemcpyHostToDevice);
}

double cVecR4::operator()(int i) {
	hipMemcpy(h_ptr, d_ptr, 4 * sizeof(double), hipMemcpyDeviceToHost);
	return h_ptr[i];
}

void cVecR4::send2Device() {
	hipMemcpy(d_ptr, h_ptr, 4 * sizeof(double), hipMemcpyHostToDevice);
}

void cVecR4::call2Host() {
	hipMemcpy(h_ptr, d_ptr, 4 * sizeof(double), hipMemcpyDeviceToHost);
}
